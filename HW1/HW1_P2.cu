
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel(uchar3 * inPixels, int width, int height, 
		float * filter, int filterWidth, 
		uchar3 * outPixels)
{
	// TODO
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int x[81] ={-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4,
				-4,-3,-2,-1, 0, 1, 2, 3, 4};
	int y[81] ={-4,-4,-4,-4,-4,-4,-4,-4,-4,
				-3,-3,-3,-3,-3,-3,-3,-3,-3,
				-2,-2,-2,-2,-2,-2,-2,-2,-2,
				-1,-1,-1,-1,-1,-1,-1,-1,-1,
				 0, 0, 0, 0, 0, 0, 0, 0, 0,
				 1, 1, 1, 1, 1, 1, 1, 1, 1,
				 2, 2, 2, 2, 2, 2, 2, 2, 2,
				 3, 3, 3, 3, 3, 3, 3, 3, 3,
				 4, 4, 4, 4, 4, 4, 4, 4, 4};
	if (col < width && row < height){
		int i = row * width + col;
		float ox, oy, oz;
		ox = oy = oz = 0;
		for(int f = 0; f < filterWidth * filterWidth; f++){
			int frow = row + y[f];
			frow = frow < 0 ? 0 : frow > height - 1 ? height - 1 : frow;
			int fcol = col + x[f];
			fcol = fcol < 0 ? 0 : fcol > width - 1 ? width - 1 : fcol;
			int p = frow * width + fcol;
			ox += inPixels[p].x * filter[f];
			oy += inPixels[p].y * filter[f];
			oz += inPixels[p].z * filter[f];
		}
		outPixels[i].x = ox;
		outPixels[i].y = oy;
		outPixels[i].z = oz;
	}
}

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		int x[81] ={-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4,
					-4,-3,-2,-1, 0, 1, 2, 3, 4};
		int y[81] ={-4,-4,-4,-4,-4,-4,-4,-4,-4,
					-3,-3,-3,-3,-3,-3,-3,-3,-3,
					-2,-2,-2,-2,-2,-2,-2,-2,-2,
					-1,-1,-1,-1,-1,-1,-1,-1,-1,
					 0, 0, 0, 0, 0, 0, 0, 0, 0,
					 1, 1, 1, 1, 1, 1, 1, 1, 1,
					 2, 2, 2, 2, 2, 2, 2, 2, 2,
					 3, 3, 3, 3, 3, 3, 3, 3, 3,
					 4, 4, 4, 4, 4, 4, 4, 4, 4};
		// TODO
		for(int row = 0; row < height; row++){
			for(int col = 0; col < width; col++){
				int i = row * width + col;
				float ox, oy, oz;
				ox = oy = oz = 0;
				for(int f = 0; f < filterWidth * filterWidth; f++){
					int frow = row + y[f];
					frow = frow < 0 ? 0 : frow > height - 1 ? height - 1 : frow;
					int fcol = col + x[f];
					fcol = fcol < 0 ? 0 : fcol > width - 1 ? width - 1 : fcol;
					int p = frow * width + fcol;
					ox += inPixels[p].x * filter[f];
					oy += inPixels[p].y * filter[f];
					oz += inPixels[p].z * filter[f];
				}
				outPixels[i].x = ox;
				outPixels[i].y = oy;
				outPixels[i].z = oz;
			}
		}
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO
		uchar3 *d_in, *d_out;
		float *fil;
		size_t nBytesImg = width * height * sizeof(uchar3) * 3;
		size_t nBytesFilter = filterWidth * filterWidth * sizeof(float);
		CHECK(hipMalloc(&d_in, nBytesImg));
		CHECK(hipMalloc(&d_out, nBytesImg));
		CHECK(hipMalloc(&fil, nBytesFilter));

        CHECK(hipMemcpy(d_in, inPixels, nBytesImg, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(fil, filter, nBytesFilter, hipMemcpyHostToDevice));

		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		blurImgKernel<<<gridSize,blockSize>>>(d_in,width,height,fil,filterWidth,d_out);

		CHECK(hipMemcpy(outPixels, d_out, nBytesImg, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_in));
        CHECK(hipFree(d_out));
		CHECK(hipFree(fil));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect 
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);
	
	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}  
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
